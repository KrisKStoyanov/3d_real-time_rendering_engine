#include "hip/hip_runtime.h"
#include "CUDAContextScheduler.cuh"

namespace HC {
	__global__ void k_Render(vec3* frameBuffer, const int areaW, const int areaH,
		vec3 rayOrigin) {
		//Thread ID offset by CTA ID with blockdim number of threads inside the grid 
		int tidX = threadIdx.x + blockIdx.x * blockDim.x;
		int tidY = threadIdx.y + blockIdx.y * blockDim.y;
		if ((tidX >= areaW) || (tidY >= areaH)) {
			return;
		}
		int pId = tidY * areaW + tidX;
		float u = (float(tidX) / float(areaW));
		float v = (float(tidY) / float(areaH));
		ray r(rayOrigin, vec3(u , v , 0.0f));
		frameBuffer[pId] = d_color(r);
	}

	__host__ void ScheduleRenderKernel(int areaW, int areaH) {

		dim3 CTAsize(8, 8);
		dim3 gridSize(1280 / CTAsize.x + 1, 720 / CTAsize.y + 1);

		int nPixels = areaW * areaH;
		size_t fBufSize = nPixels * sizeof(vec3);

		vec3* h_FBuf = (vec3*)(malloc(fBufSize));
		vec3* d_FBuf;
		
		vec3 rayOrigin = vec3(areaW / 2, areaH / 2, 0.0f);

		hipStream_t d_Stream;
		ProfileCUDA(hipStreamCreate(&d_Stream));

		int nDevices;
		ProfileCUDA(hipGetDeviceCount(&nDevices));
		const int n = nDevices;
		hipDeviceProp_t* dProps = new hipDeviceProp_t[n];
		int asyncEngines = 0;
#if defined(_DEBUG)
		std::string dPropsStr = "Devices:\n-------";
#endif
		for (int i = 0; i < nDevices; ++i) {
			dProps[i] = QueryDeviceProperties(i);
#if defined(_DEBUG)
			dPropsStr.append(
				"\nDevice ID: " + i +
				std::string("\nDevice Name: ") + dProps[i].name +
				"\nMemory Clock Rate (KHz): " + std::to_string(dProps[i].memoryClockRate) +
				"\nMemory Bus Width (bits): " + std::to_string(dProps[i].memoryBusWidth) +
				"\nPeak Memory Bandwith (GB/s): " + std::to_string(2.0 * dProps[i].memoryClockRate * (dProps[i].memoryBusWidth / 8) / 1.0e6)
			);
#endif
			if (dProps[i].asyncEngineCount > asyncEngines) {
				asyncEngines = dProps[i].asyncEngineCount;
			}
		}
#if defined(_DEBUG)
		StreamOutputToConsole(dPropsStr.c_str());
#endif
		if (asyncEngines) {
			ProfileCUDA(hipHostMalloc((void**)&d_FBuf, fBufSize));
		}
		else {
			ProfileCUDA(hipMalloc((void**)&d_FBuf, fBufSize));
		}

#if defined(_DEBUG)
		hipEvent_t startK, stopK;
		ProfileCUDA(hipEventCreate(&startK));
		ProfileCUDA(hipEventCreate(&stopK));
		ProfileCUDA(hipEventRecord(startK));
#endif
		k_Render << <gridSize, CTAsize, 0, d_Stream >> > (d_FBuf, areaW, areaH, rayOrigin);
#if defined(_DEBUG)
		ProfileCUDA(hipEventRecord(stopK));
		ProfileCUDA(hipGetLastError());
		ProfileCUDA(hipEventSynchronize(stopK));
		float kExecMs;
		ProfileCUDA(hipEventElapsedTime(&kExecMs, startK, stopK));
		ProfileCUDA(hipEventDestroy(startK));
		ProfileCUDA(hipEventDestroy(stopK));
		int actThreads = gridSize.x * CTAsize.x * areaW + gridSize.y * CTAsize.y * areaH;
		float efBw = ComputeSPEffectiveBandwith(actThreads, kExecMs);
		float compThr = ComputeComputationalThroughput(18, actThreads, kExecMs/1000);
#endif

#if defined(_DEBUG)
		hipEvent_t startDMalloc, stopDMalloc;
		ProfileCUDA(hipEventCreate(&startDMalloc));
		ProfileCUDA(hipEventCreate(&stopDMalloc));
		ProfileCUDA(hipEventRecord(startDMalloc));
#endif
		ProfileCUDA(hipMemcpyAsync(h_FBuf, d_FBuf, fBufSize, hipMemcpyDeviceToHost, d_Stream));
#if defined(_DEBUG)
		ProfileCUDA(hipEventRecord(stopDMalloc));
		ProfileCUDA(hipEventSynchronize(stopDMalloc));
		float mallocElapsedMs;
		ProfileCUDA(hipEventElapsedTime(&mallocElapsedMs, startDMalloc, stopDMalloc));
		ProfileCUDA(hipEventDestroy(startDMalloc));
		ProfileCUDA(hipEventDestroy(stopDMalloc));
		float dthBw = ComputeDeviceToHostBandwith(sizeof(float), mallocElapsedMs);
		GetPerformanceMetrics(&kExecMs, &efBw, &compThr, NULL, &dthBw);
#endif
		ProfileCUDA(hipStreamDestroy(d_Stream));

		std::ofstream ofs("./cudaRaytraceGfx.ppm", std::ios::out | std::ios::binary);
		ofs << "P6\n" << areaW << " " << areaH << "\n255\n";
		for (int yOffset = areaH - 1; yOffset >= 0; --yOffset) {
			for (int xOffset = 0; xOffset < areaW; ++xOffset) {
				size_t pixelId = yOffset * areaW + xOffset;
				vec3 v = h_FBuf[pixelId];
				int r = int(255.99f * v.r());
				int g = int(255.99f * v.g());
				int b = int(255.99f * v.b());
				std::cout << r << " " << g << " " << b << "\n";
				ofs << (unsigned char)r << (unsigned char)g << (unsigned char)b;
			}
		}
		ofs.close();

		free(h_FBuf);

		if (asyncEngines) {
			ProfileCUDA(hipHostFree(d_FBuf));
		}
		else {
			ProfileCUDA(hipFree(d_FBuf));
		}
	}

	__host__ __device__ void CheckError(hipError_t result, char const* const func, const char* const file, int const line) {
#if defined(_DEBUG)
		if (result) {
			unsigned int errId = static_cast<unsigned int>(result);
			const char* errName = hipGetErrorName(result);
			const char* errDesc = hipGetErrorString(result);
			std::string errStr =
				std::string("CUDA Error: ") + std::to_string(errId) + "\n" +
				std::string(errName) + ": " + std::string(errDesc) +
				std::string("\nFile: ") + file +
				std::string("\nLine: ") + std::to_string(line);
			
			hipError_t resetErr = hipDeviceReset();
			if (resetErr) {
				std::string resetErrStr =
					std::string("CUDA Reset Error: ") + std::to_string(errId) + "\n" +
					std::string(errName) + ": " + std::string(errDesc) +
					std::string("\nFile: ") + file +
					std::string("\nLine: ") + std::to_string(line);
				errStr.append(resetErrStr);
			}
			StreamOutputToConsole(errStr.c_str(), 3000, stderr);
			exit(99);
		}
#endif
	}

	__host__ hipDeviceProp_t QueryDeviceProperties(int dIndex) {
		hipDeviceProp_t dProps;
		ProfileCUDA(hipGetDeviceProperties(&dProps, dIndex));
		return dProps;
	}

	__host__ float ComputeSPEffectiveBandwith(int actThr, float kExecMs)
	{
		return (actThr * sizeof(float) * 3 / kExecMs / 1e6);
	}

	__host__ float ComputeComputationalThroughput(int nFlops, int actThr, float kExecS)
	{
		return (nFlops * actThr / (kExecS * 1e9));
	}

	__host__ float ComputeHostToDeviceBandwith(unsigned int bytes, float elpsdMs)
	{
		return (bytes * 1e6 / elpsdMs);
	}

	__host__ float ComputeDeviceToHostBandwith(unsigned int bytes, float elpsdMs)
	{
		return (bytes * 1e6 / elpsdMs);
	}

	__host__ std::string GetPerformanceMetrics(
		float* kExecMs, 
		float* efBw, 
		float* compThr, 
		float* htdBw, 
		float* dthBw)
	{
		std::string perfStr;
		
		if (kExecMs) {
			std::string kExecStr = "Kernel Execution Speed (MS): " + std::to_string(*kExecMs);
			perfStr.append(kExecStr);
		}

		if (efBw) {
			std::string efBwStr = "\nEffective Bandwith (GB/s): " + std::to_string(*efBw);
			perfStr.append(efBwStr);
		}

		if (compThr) {
			std::string compThrStr = "\nComputation Throughput (FLOPS/s): " + std::to_string(*compThr);
			perfStr.append(compThrStr);
		}

		if (htdBw) {
			std::string htdBwStr = "\nHost to Device bandwith (GB/s): " + std::to_string(*htdBw);
			perfStr.append(htdBwStr);
		}

		if (dthBw) {
			std::string dthBwStr = "\nDevice to Host bandwith (GB/s): " + std::to_string(*dthBw);
			perfStr.append(dthBwStr);
		}
		
		StreamOutputToConsole(perfStr.c_str(), 3000);
		return perfStr;
	}
}