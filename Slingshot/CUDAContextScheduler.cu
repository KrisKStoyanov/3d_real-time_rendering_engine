#include "hip/hip_runtime.h"
#include "CUDAContextScheduler.cuh"

namespace HC {
	__global__ void k_Render(vec3* frameBuffer, const int areaW, const int areaH,
		vec3 rayOrigin) {
		//Thread ID offset by CTA ID with blockdim number of threads inside the grid 
		int tidX = threadIdx.x + blockIdx.x * blockDim.x;
		int tidY = threadIdx.y + blockIdx.y * blockDim.y;
		if ((tidX >= areaW) || (tidY >= areaH)) {
			return;
		}
		int pId = tidY * areaW + tidX;
		float u = (float(tidX) / float(areaW));
		float v = (float(tidY) / float(areaH));
		ray r(rayOrigin, vec3(u , v , 0.0f));
		frameBuffer[pId] = d_color(r);
	}

	__host__ void ScheduleRenderKernel(int areaW, int areaH) {

		dim3 CTAsize(8, 8);
		dim3 gridSize(1280 / CTAsize.x + 1, 720 / CTAsize.y + 1);

		int nPixels = areaW * areaH;
		size_t fBufSize = nPixels * sizeof(vec3);

		vec3* h_FBuf = (vec3*)(malloc(fBufSize));
		vec3* d_FBuf;
		
		vec3 rayOrigin = vec3(areaW / 2, areaH / 2, 0.0f);

		ProfileCUDA(hipMalloc((void**)&d_FBuf, fBufSize));
		//ProfileCUDA(hipHostMalloc((void**)&d_FBuf, fBufSize));

#if defined(_DEBUG)
		QueryDeviceProperties();
		hipEvent_t startK, stopK;
		ProfileCUDA(hipEventCreate(&startK));
		ProfileCUDA(hipEventCreate(&stopK));
		ProfileCUDA(hipEventRecord(startK));
#endif
		k_Render << <gridSize, CTAsize >> > (d_FBuf, areaW, areaH, rayOrigin);
#if defined(_DEBUG)
		ProfileCUDA(hipEventRecord(stopK));
		ProfileCUDA(hipGetLastError());
		ProfileCUDA(hipEventSynchronize(stopK));
		float kExecMs;
		ProfileCUDA(hipEventElapsedTime(&kExecMs, startK, stopK));
		ProfileCUDA(hipEventDestroy(startK));
		ProfileCUDA(hipEventDestroy(stopK));
		int actThreads = gridSize.x * CTAsize.x * areaW + gridSize.y * CTAsize.y * areaH;
		float efBw = ComputeSPEffectiveBandwith(actThreads, kExecMs);
		float compThr = ComputeComputationalThroughput(18, actThreads, kExecMs/1000);
#endif

#if defined(_DEBUG)
		hipEvent_t startDMalloc, stopDMalloc;
		ProfileCUDA(hipEventCreate(&startDMalloc));
		ProfileCUDA(hipEventCreate(&stopDMalloc));
		ProfileCUDA(hipEventRecord(startDMalloc));
#endif
		ProfileCUDA(hipMemcpy(h_FBuf, d_FBuf, fBufSize, hipMemcpyDeviceToHost));
#if defined(_DEBUG)
		ProfileCUDA(hipEventRecord(stopDMalloc));
		ProfileCUDA(hipEventSynchronize(stopDMalloc));
		float mallocElapsedMs;
		ProfileCUDA(hipEventElapsedTime(&mallocElapsedMs, startDMalloc, stopDMalloc));
		ProfileCUDA(hipEventDestroy(startDMalloc));
		ProfileCUDA(hipEventDestroy(stopDMalloc));
		float dthBw = ComputeDeviceToHostBandwith(sizeof(float), mallocElapsedMs);
		GetPerformanceMetrics(kExecMs, efBw, compThr, 0.0f, dthBw);
#endif

		std::ofstream ofs("./cudaRaytraceGfx.ppm", std::ios::out | std::ios::binary);
		ofs << "P6\n" << areaW << " " << areaH << "\n255\n";
		for (int yOffset = areaH - 1; yOffset >= 0; --yOffset) {
			for (int xOffset = 0; xOffset < areaW; ++xOffset) {
				size_t pixelId = yOffset * areaW + xOffset;
				vec3 v = h_FBuf[pixelId];
				int r = int(255.99f * v.r());
				int g = int(255.99f * v.g());
				int b = int(255.99f * v.b());
				std::cout << r << " " << g << " " << b << "\n";
				ofs << (unsigned char)r << (unsigned char)g << (unsigned char)b;
			}
		}
		ofs.close();

		free(h_FBuf);
		ProfileCUDA(hipFree(d_FBuf));
		//ProfileCUDA(hipHostFree(d_FBuf));
	}

	__host__ __device__ void CheckError(hipError_t result, char const* const func, const char* const file, int const line) {
#if defined(_DEBUG)
		if (result) {
			unsigned int errId = static_cast<unsigned int>(result);
			const char* errName = hipGetErrorName(result);
			const char* errDesc = hipGetErrorString(result);
			std::string errStr =
				std::string("CUDA Error: ") + std::to_string(errId) + "\n" +
				std::string(errName) + ": " + std::string(errDesc) +
				std::string("\nFile: ") + file +
				std::string("\nLine: ") + std::to_string(line);
			
			hipError_t resetErr = hipDeviceReset();
			if (resetErr) {
				std::string resetErrStr =
					std::string("CUDA Reset Error: ") + std::to_string(errId) + "\n" +
					std::string(errName) + ": " + std::string(errDesc) +
					std::string("\nFile: ") + file +
					std::string("\nLine: ") + std::to_string(line);
				errStr.append(resetErrStr);
			}
			StreamOutputToConsole(errStr.c_str(), 3000, stderr);
			exit(99);
		}
#endif
	}

	__host__ std::string QueryDeviceProperties() {
		int nDevices;
		ProfileCUDA(hipGetDeviceCount(&nDevices));
		const int n = nDevices;
		hipDeviceProp_t* dProps = new hipDeviceProp_t[n];
		std::string dPropsStr = "Devices:\n-------";
		for (int i = 0; i < nDevices; ++i) {
			ProfileCUDA(hipGetDeviceProperties(&dProps[i], i));
			dPropsStr.append(
				"\nDevice ID: " + i +
				std::string("\nDevice Name: ") + dProps[i].name +
				"\nMemory Clock Rate (KHz): " + std::to_string(dProps[i].memoryClockRate) +
				"\nMemory Bus Width (bits): " + std::to_string(dProps[i].memoryBusWidth) +
				"\nPeak Memory Bandwith (GB/s): " + std::to_string(2.0 * dProps[i].memoryClockRate * (dProps[i].memoryBusWidth / 8) / 1.0e6)
			);
		}
		StreamOutputToConsole(dPropsStr.c_str());
		return dPropsStr;
	}

	__host__ float ComputeSPEffectiveBandwith(int actThr, float kExecMs)
	{
		return (actThr * sizeof(float) * 3 / kExecMs / 1e6);
	}

	__host__ float ComputeComputationalThroughput(int nFlops, int actThr, float kExecS)
	{
		return (nFlops * actThr / (kExecS * 1e9));
	}

	__host__ float ComputeHostToDeviceBandwith(unsigned int bytes, float elpsdMs)
	{
		return (bytes * 1e6 / elpsdMs);
	}

	__host__ float ComputeDeviceToHostBandwith(unsigned int bytes, float elpsdMs)
	{
		return (bytes * 1e6 / elpsdMs);
	}

	__host__ std::string GetPerformanceMetrics(float kExecMs, float efBw, float compThr, float htdBw, float dthBw)
	{
		std::string perfStr;
		
		if (kExecMs > 0.0f) {
			std::string kExecStr = "Kernel Execution Speed (MS): " + std::to_string(kExecMs);
			perfStr.append(kExecStr);
		}

		if (efBw > 0.0f) {
			std::string efBwStr = "\nEffective Bandwith (GB/s): " + std::to_string(efBw);
			perfStr.append(efBwStr);
		}

		if (compThr > 0.0f) {
			std::string compThrStr = "\nComputation Throughput (FLOPS/s): " + std::to_string(compThr);
			perfStr.append(compThrStr);
		}

		if (htdBw > 0.0f) {
			std::string htdBwStr = "\nHost to Device bandwith (GB/s): " + std::to_string(htdBw);
			perfStr.append(htdBwStr);
		}

		if (dthBw > 0.0f) {
			std::string dthBwStr = "\nDevice to Host bandwith (GB/s): " + std::to_string(dthBw);
			perfStr.append(dthBwStr);
		}
		
		StreamOutputToConsole(perfStr.c_str(), 3000);
		return perfStr;
	}
}