#include "hip/hip_runtime.h"
#include "CUDAContextScheduler.cuh"

namespace HC {
	__global__ void k_Render(vec3* frameBuffer, int nPixels, int areaW, int areaH) {
		
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid > nPixels) {
			return;
		}
		int pixelY = areaH - tid / areaW;
		int pixelX = tid - (tid / areaW) * areaW;
		float r = (float)pixelX / (float)areaW;;
		float g = (float)pixelY / (float)areaH;
		float b = 0.2f;
		vec3 v(r, g, b);
		frameBuffer[tid] = v;
	}

	//Pending compute concurrency implementation through CUDA streams (local async engines = 6)

	__host__ bool InvokeRenderKernel(vec3*& screenBuffer, int areaW, int areaH) {

		int nPixels = areaW * areaH;

		//dim3 CTAsize(8, 8);
		//dim3 gridSize(areaW / CTAsize.x + 1, areaH / CTAsize.y + 1);

		int CTASize = 64;
		int gridSize = nPixels / CTASize + 1;

		size_t fBufSize = nPixels * sizeof(vec3);

		screenBuffer = (vec3*)(malloc(fBufSize));
		vec3* d_FBuf;
		
		vec3 rayOrigin = vec3(areaW / 2, areaH / 2, 0.0f);
		
		int nDevices;
		ProfileCUDA(hipGetDeviceCount(&nDevices));
		const int n = nDevices;
		hipDeviceProp_t* dProps = new hipDeviceProp_t[n];
		int asyncEngines = 0;
#if defined(_DEBUG)
		std::string dPropsStr = "Devices:\n-------";
#endif
		for (int i = 0; i < nDevices; ++i) {
			dProps[i] = QueryDeviceProperties(i);
#if defined(_DEBUG)
			dPropsStr.append(
				"\nDevice ID: " + i +
				std::string("\nDevice Name: ") + dProps[i].name +
				"\nMemory Clock Rate (KHz): " + std::to_string(dProps[i].memoryClockRate) +
				"\nMemory Bus Width (bits): " + std::to_string(dProps[i].memoryBusWidth) +
				"\nPeak Memory Bandwith (GB/s): " + std::to_string(2.0 * dProps[i].memoryClockRate * (dProps[i].memoryBusWidth / 8) / 1.0e6)
			);
#endif
			if (dProps[i].asyncEngineCount > asyncEngines) {
				asyncEngines = dProps[i].asyncEngineCount;
			}
		}
#if defined(_DEBUG)
		StreamOutputToConsole(dPropsStr.c_str());
#endif

		//Experimental: (override asyncEngines to 0 to return to default implementation)
		asyncEngines = 0;
		//----------------
		if (asyncEngines) {
			const int nEngines = asyncEngines;
			hipStream_t* d_Streams = new hipStream_t[nEngines];
			
			size_t d_FbufFeatureSize = fBufSize / nEngines;

			for (int i = 0; i < nEngines; ++i) {
				ProfileCUDA(hipStreamCreate(&d_Streams[i]));
			}
			
			ProfileCUDA(hipHostMalloc((void**)&d_FBuf, fBufSize));

#if defined(_DEBUG)
			hipEvent_t startK, stopK;
			ProfileCUDA(hipEventCreate(&startK));
			ProfileCUDA(hipEventCreate(&stopK));
			ProfileCUDA(hipEventRecord(startK));
#endif

			for (int i = 0; i < nEngines; ++i) {
				int wOffset ;
				int hOffset;
				k_Render << <gridSize, CTASize, 0, d_Streams[i] >> > 
					(d_FBuf, nPixels, areaW, areaH);
			}
			
#if defined(_DEBUG)
			ProfileCUDA(hipEventRecord(stopK));
			ProfileCUDA(hipGetLastError());
			ProfileCUDA(hipEventSynchronize(stopK));
			float kExecMs;
			ProfileCUDA(hipEventElapsedTime(&kExecMs, startK, stopK));
			ProfileCUDA(hipEventDestroy(startK));
			ProfileCUDA(hipEventDestroy(stopK));
			float efBw = ComputeSPEffectiveBandwith(nPixels, kExecMs);
			float compThr = ComputeComputationalThroughput(18, nPixels, kExecMs / 1000);
#endif

#if defined(_DEBUG)
			hipEvent_t startDMalloc, stopDMalloc;
			ProfileCUDA(hipEventCreate(&startDMalloc));
			ProfileCUDA(hipEventCreate(&stopDMalloc));
			ProfileCUDA(hipEventRecord(startDMalloc));
#endif
			for (int i = 0; i < nEngines; ++i) {
				ProfileCUDA(hipMemcpyAsync(screenBuffer, d_FBuf, fBufSize, hipMemcpyDeviceToHost, d_Streams[i]));
			}
#if defined(_DEBUG)
			ProfileCUDA(hipEventRecord(stopDMalloc));
			ProfileCUDA(hipEventSynchronize(stopDMalloc));
			float mallocElapsedMs;
			ProfileCUDA(hipEventElapsedTime(&mallocElapsedMs, startDMalloc, stopDMalloc));
			ProfileCUDA(hipEventDestroy(startDMalloc));
			ProfileCUDA(hipEventDestroy(stopDMalloc));
			float dthBw = ComputeDeviceToHostBandwith(sizeof(float), mallocElapsedMs);
			GetPerformanceMetrics(&kExecMs, &efBw, &compThr, NULL, &dthBw);
#endif
			for (int i = 0; i < nEngines; ++i) {
				ProfileCUDA(hipStreamDestroy(d_Streams[i]));
			}
		}
		//----------------
		//Default:
		//----------------
		else {
			ProfileCUDA(hipMalloc((void**)&d_FBuf, fBufSize));

#if defined(_DEBUG)
			hipEvent_t startK, stopK;
			ProfileCUDA(hipEventCreate(&startK));
			ProfileCUDA(hipEventCreate(&stopK));
			ProfileCUDA(hipEventRecord(startK));
#endif
			k_Render << <gridSize, CTASize, 0, 0 >> > (d_FBuf, nPixels, areaW, areaH);
#if defined(_DEBUG)
			ProfileCUDA(hipEventRecord(stopK));
			ProfileCUDA(hipGetLastError());
			ProfileCUDA(hipEventSynchronize(stopK));
			float kExecMs;
			ProfileCUDA(hipEventElapsedTime(&kExecMs, startK, stopK));
			ProfileCUDA(hipEventDestroy(startK));
			ProfileCUDA(hipEventDestroy(stopK));
			float efBw = ComputeSPEffectiveBandwith(nPixels, kExecMs);
			float compThr = ComputeComputationalThroughput(18, nPixels, kExecMs / 1000);
#endif

#if defined(_DEBUG)
			hipEvent_t startDMalloc, stopDMalloc;
			ProfileCUDA(hipEventCreate(&startDMalloc));
			ProfileCUDA(hipEventCreate(&stopDMalloc));
			ProfileCUDA(hipEventRecord(startDMalloc));
#endif
			ProfileCUDA(hipMemcpyAsync(screenBuffer, d_FBuf, fBufSize, hipMemcpyDeviceToHost));
#if defined(_DEBUG)
			ProfileCUDA(hipEventRecord(stopDMalloc));
			ProfileCUDA(hipEventSynchronize(stopDMalloc));
			float mallocElapsedMs;
			ProfileCUDA(hipEventElapsedTime(&mallocElapsedMs, startDMalloc, stopDMalloc));
			ProfileCUDA(hipEventDestroy(startDMalloc));
			ProfileCUDA(hipEventDestroy(stopDMalloc));
			float dthBw = ComputeDeviceToHostBandwith(sizeof(float), mallocElapsedMs);
			GetPerformanceMetrics(&kExecMs, &efBw, &compThr, NULL, &dthBw);
#endif
		}
		//----------------

		GenPPMFile("GfxExp", screenBuffer, areaW, areaH);

		//free(screenBuffer);

		if (asyncEngines) {
			ProfileCUDA(hipHostFree(d_FBuf));
		}
		else {
			ProfileCUDA(hipFree(d_FBuf));
		}

		return true;
	}

	__host__ __device__ void CheckError(hipError_t result, char const* const func, const char* const file, int const line) {
#if defined(_DEBUG)
		if (result) {
			unsigned int errId = static_cast<unsigned int>(result);
			const char* errName = hipGetErrorName(result);
			const char* errDesc = hipGetErrorString(result);
			std::string errStr =
				std::string("CUDA Error: ") + std::to_string(errId) + "\n" +
				std::string(errName) + ": " + std::string(errDesc) +
				std::string("\nFile: ") + file +
				std::string("\nLine: ") + std::to_string(line);
			
			hipError_t resetErr = hipDeviceReset();
			if (resetErr) {
				std::string resetErrStr =
					std::string("CUDA Reset Error: ") + std::to_string(errId) + "\n" +
					std::string(errName) + ": " + std::string(errDesc) +
					std::string("\nFile: ") + file +
					std::string("\nLine: ") + std::to_string(line);
				errStr.append(resetErrStr);
			}
			StreamOutputToConsole(errStr.c_str(), 3000, stderr);
			exit(99);
		}
#endif
	}

	__host__ hipDeviceProp_t QueryDeviceProperties(int dIndex) {
		hipDeviceProp_t dProps;
		ProfileCUDA(hipGetDeviceProperties(&dProps, dIndex));
		return dProps;
	}

	__host__ float ComputeSPEffectiveBandwith(int actThr, float kExecMs)
	{
		return (actThr * sizeof(float) * 3 / kExecMs / 1e6);
	}

	__host__ float ComputeComputationalThroughput(int nFlops, int actThr, float kExecS)
	{
		return (nFlops * actThr / (kExecS * 1e9));
	}

	__host__ float ComputeHostToDeviceBandwith(unsigned int bytes, float elpsdMs)
	{
		return (bytes * 1e6 / elpsdMs);
	}

	__host__ float ComputeDeviceToHostBandwith(unsigned int bytes, float elpsdMs)
	{
		return (bytes * 1e6 / elpsdMs);
	}

	__host__ std::string GetPerformanceMetrics(
		float* kExecMs, 
		float* efBw, 
		float* compThr, 
		float* htdBw, 
		float* dthBw,
		unsigned int conSleepMs)
	{
		std::string perfStr;
		
		if (kExecMs) {
			std::string kExecStr = "Kernel Execution Speed (MS): " + std::to_string(*kExecMs);
			perfStr.append(kExecStr);
		}

		if (efBw) {
			std::string efBwStr = "\nEffective Bandwith (GB/s): " + std::to_string(*efBw);
			perfStr.append(efBwStr);
		}

		if (compThr) {
			std::string compThrStr = "\nComputation Throughput (FLOPS/s): " + std::to_string(*compThr);
			perfStr.append(compThrStr);
		}

		if (htdBw) {
			std::string htdBwStr = "\nHost to Device bandwith (GB/s): " + std::to_string(*htdBw);
			perfStr.append(htdBwStr);
		}

		if (dthBw) {
			std::string dthBwStr = "\nDevice to Host bandwith (GB/s): " + std::to_string(*dthBw);
			perfStr.append(dthBwStr);
		}
		
		StreamOutputToConsole(perfStr.c_str(), conSleepMs);
		return perfStr;
	}

	__host__ void GenPPMFile(const char* fileName, vec3* buffer, const int imgW, const int imgH) {
		std::string fn = std::string("./") + fileName + ".ppm";
		std::ofstream ofsGpu(fn.c_str(), std::ios::out | std::ios::binary);
		ofsGpu << "P6\n" << imgW << " " << imgH << "\n255\n";
		int nPixels = imgW * imgH;
		for (int i = 0; i < nPixels; ++i) {
			vec3 v = buffer[i];
			int r = int(255.99f * v.r());
			int g = int(255.99f * v.g());
			int b = int(255.99f * v.b());
			ofsGpu << (unsigned char)r << (unsigned char)g << (unsigned char)b;
		}
		ofsGpu.close();
	}
}